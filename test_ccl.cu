#include "test_ccl.h"
 
 test_ccl::test_ccl(unsigned int size){
 	if (size%32)
 	{
 		LOG(LOG_ERROR)<<"size not an multipel of 32.  size is: "<<size;
 		exit(2);
 	}
 	L=size;
 	hipMalloc((void**)&ImageD, sizeof(int)*L*L);
 	hipMalloc((void**)&LabelD, sizeof(int)*L*L);
 	ImageH = new int [L*L];
 }

 test_ccl::~test_ccl(){
 	hipFree(ImageD);
 	hipFree(LabelD);
 }

 void test_ccl::RandomImage(double p){
 	std::random_device device;
    std::mt19937 gen(device());
    std::uniform_real_distribution dist(0,1);
    for (int i = 0; i < L*L; ++i)
     {
     	ImageH[i]=(p<=dist(gen))?0:1;
     } 
     hipMemcpy(ImageD, ImageH, sizeof(int)*L*L, hipMemcpyHostToDevice);

  }

 void test_ccl::Labeing(){
 	
 }