#include "test_ccl.h"
#include "gpu_kernal.h"
#include <iostream>
// #include <hip/hip_runtime.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
		LOG(LOG_ERROR)<<"GPUassert: "<< hipGetErrorString(code)<<" "<<file<<" "<<line;
	   if (abort) exit(code);
   }
}

 test_ccl::test_ccl(unsigned int size){
 	if (size%32)
 	{
 		LOG(LOG_ERROR)<<"size not an multipel of 32.  size is: "<<size;
 		exit(2);
 	}
 	L=size;
 	hipMalloc((void**)&ImageD, sizeof(int)*L*L);
	hipMalloc((void**)&LabelD, sizeof(int)*L*L);
	hipMalloc((void**)&SizeD, sizeof(int)*L*L);
	ImageH = new int [L*L];
	gpuErrchk(hipPeekAtLastError());
 
 }

 test_ccl::~test_ccl(){
	hipDeviceSynchronize();
 	hipFree(ImageD);
	hipFree(LabelD);
	hipFree(SizeD);
	
	gpuErrchk(hipPeekAtLastError());

 }

 void test_ccl::RandomImage(double p){
 	std::random_device device;
    std::mt19937 gen(device());
    std::uniform_real_distribution<> dist(0,1);
    for (int i = 0; i < L*L; ++i)
     {
     	ImageH[i]=(p<=dist(gen))?1:0;
	 } 
	 hipMemcpy(ImageD, ImageH, sizeof(int)*L*L, hipMemcpyHostToDevice);
	 gpuErrchk(hipPeekAtLastError());


  }

 void test_ccl::Labeling(){
	dim3 perBlock1(32,32);
	dim3 numBlock1(1,L/32);
	HA4_Strip_Labeling<<<numBlock1,perBlock1,sizeof(int)*L/32>>>(ImageD,LabelD,L);
	dim3 perBlock2(32,(L/32)<32?L/32:32);
	dim3 numBlock2(L/32,(L/32)<32?1:L/64);
	HA4_Strip_Merge<<<numBlock2,perBlock2>>>(ImageD,LabelD,L,32);
	dim3 perBlock3(32,32);
	dim3 numBlock3(L/32,L/32);
	HA4_Relabeling<<<numBlock3,perBlock3>>>(ImageD,LabelD,L);
	
	// hipDeviceSynchronize();
	// gpuErrchk(hipPeekAtLastError());
 	
 }

 void test_ccl::PrintLabel(){
	 int * Label = new int [L*L];
	 hipDeviceSynchronize();

	 hipMemcpy(Label, LabelD, sizeof(int)*L*L, hipMemcpyDeviceToHost);
	 for(int y = 0;y<L;y++){
		for(int x = 0;x<L;x++){
			std::cout<<Label[y*L+x]<<" ";
		}
		std::cout<<std::endl;
	 }
	 gpuErrchk(hipPeekAtLastError());
	 delete[] Label;
	 
 }

 void test_ccl::ClusterSize(){
	hipMemset(SizeD,0,sizeof(int)*L*L);
	int *SizeH =new int [L*L];
	dim3 perBlock1(32,32);
	dim3 numBlock1(1,L/32);
	HA4_Strip_Labeling<<<numBlock1,perBlock1,sizeof(int)*L/32>>>(ImageD,LabelD,L);
	dim3 perBlock2(32,(L/32)<32?L/32:32);
	dim3 numBlock2(L/32,(L/32)<32?1:L/64);
	HA4_Strip_Merge<<<numBlock2,perBlock2>>>(ImageD,LabelD,L,32);
	dim3 perBlock3(32,32);
	dim3 numBlock3(L/32,L/32);
	HA4_ClusterSize<<<numBlock3,perBlock3>>>(ImageD,LabelD,L,SizeD);
	hipMemcpy(SizeH, SizeD, sizeof(int)*L*L, hipMemcpyDeviceToHost);
	int s_max=0;
	for(int i=0;i<L*L;i++){
		if (SizeH[i]>s_max){
			s_max=SizeH[i];
		}
	}
	std::cout<<s_max<<std::endl;
	

	 
 }