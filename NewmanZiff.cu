#include "NewmanZiff.h"
#include "gpu_kernal.h"

#include "Logging.hpp"

#include <algorithm>

#include <iostream>

#define gpuErrchk(ans)                         \
  if ((ans) != hipSuccess) {                  \
    LOG(LOG_ERROR) << hipGetErrorString(ans) <<" : "<<ans; \
  }

#define LastError() gpuErrchk(hipPeekAtLastError())

NewmanZiff::NewmanZiff(int size,int stride_):L(size),stride(stride_){
    it=0;
    randVH=new int[L*L];
    for (int i=0;i<L*L;i++){
        randVH[i]=i;
    }
    gen.seed(12345UL);
    gpuErrchk(hipMalloc((void**)&LabelD, sizeof(int)*L*L));
    gpuErrchk(hipMalloc((void**)&ImageD, sizeof(int)*L*L));
    gpuErrchk(hipMalloc((void**)&SizeD, sizeof(int)*L*L));
    gpuErrchk(hipMalloc((void**)&randVD, sizeof(int)*L*L));
}

NewmanZiff::~NewmanZiff(){
    hipDeviceSynchronize();
    hipFree(LabelD);
    hipFree(ImageD);
    hipFree(SizeD);
    hipFree(randVD);
    delete[] randVH;
}

bool NewmanZiff::nextStep(){
    if (it+stride<L*L){
        int perBlock = (stride<512)?stride:512;
        int numBlock = stride/512+1;
        SetMem2Value<<<numBlock,perBlock>>>(ImageD, randVD, it, stride);
        it+=stride;
        LastError();
        return true;
    }
    else{
        return false;
    }
}

int NewmanZiff::GetBigestCluster(){
    hipMemset(SizeD,0,sizeof(int)*L*L);
	int *SizeH =new int [L*L];
	dim3 perBlock1(32,32);
	dim3 numBlock1(1,L/32);
    HA4_Strip_Labeling<<<numBlock1,perBlock1,sizeof(int)*32>>>(ImageD,LabelD,L);
    dim3 perBlock2(32,32);
	dim3 numBlock2(L/32,L/32);
    HA4_Strip_Merge<<<numBlock2,perBlock2>>>(ImageD,LabelD,L,32);
	dim3 perBlock3(32,32);
	dim3 numBlock3(L/32,L/32);
    HA4_ClusterSize<<<numBlock3,perBlock3>>>(ImageD,LabelD,L,SizeD);
    hipDeviceSynchronize();
	hipMemcpy(SizeH, SizeD, sizeof(int)*L*L, hipMemcpyDeviceToHost);
    int s_max = 0;
    int sum = 0;

	for(int i=0;i<L*L;i++){

        sum+=SizeH[i];
		if (SizeH[i]>s_max){

			s_max=SizeH[i];
		}
    }
    if(sum!=it){
        LOG(LOG_ERROR)<< "clusterzie gone wrong  sum: "<<sum<<" it: "<<it;
    }
    LastError();
    return s_max;
}

std::vector<int> NewmanZiff::iteration(){
    std::vector<int> result;
    std::shuffle(&randVH[0], &randVH[L*L], gen);
    hipMemcpy(randVD, randVH, sizeof(int)*L*L, hipMemcpyHostToDevice);
    hipMemset(ImageD,0,sizeof(int)*L*L);
    
    while (nextStep()){
        result.push_back(GetBigestCluster());
    }
    it=0;
    LastError();
    return result;
}

void NewmanZiff::PrintLabel(){
    int * Label = new int [L*L];
    hipDeviceSynchronize();
    dim3 perBlock3(32,32);
	dim3 numBlock3(L/32,L/32);
	HA4_Relabeling<<<numBlock3,perBlock3>>>(ImageD,LabelD,L);
    hipDeviceSynchronize();

    hipMemcpy(Label, LabelD, sizeof(int)*L*L, hipMemcpyDeviceToHost);
    for(int y = 0;y<L;y++){
       for(int x = 0;x<L;x++){
           std::cout<<Label[y*L+x]<<" ";
       }
       std::cout<<"\t\t"<<y*L<<std::endl;
    }
    // gpuErrchk(hipPeekAtLastError());

    std::cout<<std::endl;
    std::cout<<std::endl;
    delete[] Label;
    
}