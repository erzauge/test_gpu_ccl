#include "hip/hip_runtime.h"
#include "gpu_kernal.h"
#include <stdio.h>

__device__ int start_distance(int p,int x){
    return __clz(~(p<<(32-x)));
}

__device__ int end_distance(int p,int x){
    return __ffs(~(p>>(x+1)));
}

__device__ void merge(int *L,int label1,int label2){
    //find
    while (label1!=label2&&label1!=L[label1]){
        label1=L[label1];
    }

    while (label1!=label2&&label2!=L[label2]){
        label2=L[label2];
    }

    
    while (label1!=label2){
        if (label1<label2) {
            //swap
            int swap = label1;
            label1 = label2;
            label2 = swap;
        }

        int label3 = atomicMin(&L[label1],label2);
        if (label1==label3){
            label1=label2;
        }
        else{
            label1=label3;
        }
    }
}


__global__ void HA4_Strip_Labeling(int *I,int *L, unsigned width) {
    extern __shared__ int sPixels[];
    int lineBase    = (blockIdx.y*blockDim.y+threadIdx.y)*width+ threadIdx.x;
    int distanceY   = 0;
    int distanceYm  = 0;
    for(int i=0; i < width;i+=32){
        int id      = lineBase+i;
        int pY      = I[id];
        int pixelsY = __ballot_sync(FULL_MASK,pY);
        int sDistY  = start_distance(pixelsY,threadIdx.x);
        if (pY && sDistY==0){
            if(threadIdx.x!=0){
                L[id]=id;
            }
            else{
                L[id]=id-distanceY;
            }
        }
        __syncthreads();
        if (threadIdx.x==0){
            sPixels[threadIdx.y]=pixelsY;
        } 
        __syncthreads();
        int pixelsYm    = (threadIdx.y>0)?sPixels[threadIdx.y-1]:0;
        int pYm         = (pixelsYm>>threadIdx.x)&1;
        int sDistYm     = start_distance(pixelsYm,threadIdx.x);
        if (threadIdx.x==0){
            sDistY  = distanceY;
            sDistYm = distanceYm;
        }
        if (pY && pYm && (sDistY==0 || sDistYm==0)){
            int label1 = id - sDistY;
            int label2 = id - width -sDistYm;
            merge(L,label1,label2);
        }
        int d       = start_distance(pixelsYm,32);
        distanceYm  = d+(d==32?distanceYm:0);
        d           = start_distance(pixelsY,32);
        distanceY   = d+(d==32?distanceY:0);
    }
}

__global__ void HA4_Strip_Merge(int *I,int *L, unsigned width,unsigned blockH){
    int y = (blockIdx.y*blockDim.y+threadIdx.y);
    int x = (blockIdx.x*blockDim.x+threadIdx.x);
    if (y>0&&y<width){
        int idY     = y*width+x;
        int idYm    = idY-width;
        int pY      = I[idY];
        int pYm     = I[idYm];
        int pixelsY = __ballot_sync(FULL_MASK,pY);
        int pixelsYm= __ballot_sync(FULL_MASK,pYm);
        if (pY && pYm){
            int sDistY  = start_distance(pixelsY,threadIdx.x);
            int sDistYm = start_distance(pixelsYm,threadIdx.x);
            if (sDistY==0 || sDistYm==0){
                merge(L,idY-sDistY,idYm-sDistYm);
            }
        }
    }
}

__global__ void HA4_Relabeling(int *I,int *L, unsigned width){
    int y = (blockIdx.y*blockDim.y+threadIdx.y);
    int x = (blockIdx.x*blockDim.x+threadIdx.x);
    int id = y*width+x;
    int p = I[id];
    int pixels = __ballot_sync(FULL_MASK,p);
    int sDist =start_distance(pixels,threadIdx.x);
    int label = 0;
    if (p && sDist==0){
        label = L[id];
        while (label != L[label]){
            label = L[label];
        }
    }
    label=__shfl_sync(FULL_MASK, label, threadIdx.x -sDist);
    if (p){
        L[id]=label;
    }
    else{
        L[id]=-1;
    }

}

__global__ void HA4_ClusterSize(int *I,int *L, unsigned width,int *S){
    int y = (blockIdx.y*blockDim.y+threadIdx.y);
    int x = (blockIdx.x*blockDim.x+threadIdx.x);
    int id = y*width+x;
    int p = I[id];
    int pixels =__ballot_sync(FULL_MASK,p);
    int sDist = start_distance(pixels,threadIdx.x);
    int count = end_distance(pixels,threadIdx.x);
    if (p && sDist==0){
        int label= L[id];
        while (label!=L[label]){
            label=L[label];
        }
        if(count==0){
            count=32-threadIdx.x;
        }
        atomicAdd(&S[label],count);
    }
}

__global__ void SetMem2Value(int * write, int * id, int start, int stride,int value){
    int x = (blockIdx.x*blockDim.x+threadIdx.x);
    if(x<stride){
        write[id[start+x]]=value;
    }
}
