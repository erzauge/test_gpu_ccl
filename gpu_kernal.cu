
#include <hip/hip_runtime.h>
#define FULL_MASK 0xffffffff

__device__ int start_distance(int p,int x){
    return __clz(~(p<<(32-x)));
}

__device__ int end_distance(int p,int x){
    return __ffs(~(p>>(x+1)));
}

__device__ void merge(int *L,int label1,int label2){
    while (label1!=label2&&label1!=L[label1]){
        label1=L[label1];
    }

    while (label1!=label2&&label2!=L[label2]){
        label2=L[label2];
    }

    while (label1!=label2){
        if (label1<label2) {
            int swap = label1;
            label2 = label1;
            label1 = swap;
        }
        int label3 = atomicMin(&L[label1],label2);
        if (label1==label3){
            label1=label2;
        }
        else{
            label1=label3;
        }
    }
    
}


__global__ void HA4_Strip_Labeling(int *I,int *L, unsigned width) {
    extern __shared__ int sPixels[];
    int lineBase    = (blockIdx.y*blockDim.y+threadIdx.y)*width+ threadIdx.x;
    int distanceY   = 0;
    int distanceYm  = 0;
    for(int i=0; i < width;i+=32){
        int id      = lineBase+i;
        int pY      = I[id];
        int pixelsY = __ballot_sync(FULL_MASK,pY);
        int sDistY  = start_distance(pixelsY,threadIdx.x);
        if (pY && sDistY==0){
            if(threadIdx.x!=0){
                L[id]=id;
            }
            else{
                L[id]=id-distanceY;
            }
        }
        if (threadIdx.x==0){
            sPixels[threadIdx.y]=pixelsY;
        } 
        __syncthreads();
        int pixelsYm    = threadIdx.y>0?sPixels[threadIdx.y-1]:0;
        int pYm         = pixelsYm&(1>>threadIdx.y);
        int sDistYm     = start_distance(pixelsYm,threadIdx.x);
        if (threadIdx.x==0){
            sDistY  = distanceY;
            sDistYm = distanceYm;
        }
        if (pY && pYm && (sDistY==0 || sDistYm==0)){
            int label1 = id - sDistY;
            int label2 = id - width -sDistYm;
            merge(L,label1,label2);
        }
        int d       = start_distance(pixelsYm,32);
        distanceYm  = d+(d==32?distanceYm:0);
        d           = start_distance(pixelsY,32);
        distanceY   = d+(d==32?distanceY:0);
    }
}

//to be removed 
int main(){

    return 0;
}
