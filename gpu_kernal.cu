
#include <hip/hip_runtime.h>
#define FULL_MASK 0xffffffff

__device__ int start_distance(int p,int x){
    return __clz(~(p<<(32-x)));
}

__device__ int end_distance(int p,int x){
    return __ffs(~(p>>(x+1)));
}

__device__ void merge(int *L,int label1,int label2){
    while (label1!=label2&&label1!=L[label1]){
        label1=L[label1];
    }

    while (label1!=label2&&label2!=L[label2]){
        label2=L[label2];
    }

    while (label1!=label2){
        if (label1<label2) {
            int swap = label1;
            label2 = label1;
            label1 = swap;
        }
        int label3 = atomicMin(&L[label1],label2);
        if (label1==label3){
            label1=label2;
        }
        else{
            label1=label3;
        }
    }
    
}


__global__ void HA4_Strip_Labeling(int *I,int *L, unsigned width) {
    int lineBase    = (blockIdx.y*blockDim.y+threadIdx.y)*width+ threadIdx.x;
    int distanceY   = 0;
    int distanceYm  = 0;
    for(int i=0; i < width;i+=32){
        int id      = lineBase+i;
        int pY      = I[id];
        int pixelsY = __ballot_sync(FULL_MASK,pY);
        int sDistY  = start_distance(pixelsY,threadIdx.x);
        if (pY && sDistY==0){
            if(threadIdx.x!=0){
                L[id]=id;
            }
            else{
                L[id]=id-distanceY;
            }
        }
        //to be continud
    }
}

//to be removed 
int main(){

    return 0;
}
