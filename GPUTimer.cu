#include "GPUTimer.h"
GPUTimer::GPUTimer()
{
    hipEventCreate(&start);
    hipEventCreate(&stop);
}

GPUTimer::~GPUTimer()
{
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

void GPUTimer::Start(){
    hipEventRecord(start,0);
}

void GPUTimer::Stop(){
    hipEventRecord(stop,0);
}

float GPUTimer::Elapsed(){
    float elapsed;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    return elapsed;
}